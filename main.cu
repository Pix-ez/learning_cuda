#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <vector>
#include <iomanip>
#include "utils.h"
#include "config.h"


#define CHECK_CUDA(call) \
    do { \
        hipError_t status = call; \
        if (status != hipSuccess) { \
            std::cerr << "CUDA error at line " << __LINE__ << ": " << hipGetErrorString(status) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at line " << __LINE__ << ": " << status << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


// Kernel declaration
__global__ void naiveMatrixMultiply(float *A, float *B, float *C, int M, int N, int K);
__global__ void tiledMatrixMultiply(float *A, float *B, float *C, int M, int N, int K);

int main(){

    //Matrix: C = A x B --> [M x K] * [K * N] = [M x N]
    int M = 8192;
    int K = 1024;
    int N = 4096;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    //Allocate host memory 
    float *h_A = (float*)malloc(size_A);
    float *h_B = (float*)malloc(size_B);
    float *h_C_naive = (float*)malloc(size_C);
    float *h_C_cpu = (float*)malloc(size_C);

    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    // Copy inputs to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    //kernel launch 
    dim3 default_blockDim(16, 16);
    dim3 default_gridDim((N + default_blockDim.x - 1) / default_blockDim.x,
                         (M + default_blockDim.y - 1) / default_blockDim.y);
   

    int iterations = 20;

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        // matmul_cpu(h_A, h_B, h_C_cpu, M, N, K);
        naiveMatrixMultiply<<<default_gridDim, default_blockDim>>>(d_A, d_B, d_C, M, N, K);
        hipDeviceSynchronize();
    }

 



    //############################################## Naive MM ##############################################
    double gpu_total_time = 0.0;


    for (int i=0; i<iterations; ++i){
        double start_time = get_time();
        naiveMatrixMultiply<<<default_gridDim, default_blockDim>>>(d_A, d_B, d_C, M, N, K);
        hipDeviceSynchronize();  // wait for kernel to finish to get correct time
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }

    double gpu_avg_time = gpu_total_time / double(iterations);
    printf("Naive MM average time: %f miliseconds\n", (gpu_avg_time * TIME_SCALE_MS));

    hipMemcpy(h_C_naive, d_C, size_C, hipMemcpyDeviceToHost);
    // compare_results(h_C, h_C_cpu, M, N);




    //############################################## Tiled kernel ##############################################
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
    
    
    gpu_total_time = 0.0;
    for (int i=0; i<iterations; ++i){
        double start_time = get_time();
        tiledMatrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
        hipDeviceSynchronize();  // wait for kernel to finish to get correct time
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }

    gpu_avg_time = gpu_total_time / double(iterations);
    
    printf("Tiled MM average time: %f miliseconds\n", (gpu_avg_time * TIME_SCALE_MS));

    // // Copy result back
    // hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);


 //###################### CPU reference #####################################################
    // double cpu_total_time = 0.0;
    // for (int i=0; i<iterations; ++i){
    //     double start_time = get_time();
    //     matmul_cpu(h_A, h_B, h_C_cpu, M, N, K);
    //     double end_time = get_time();
    //     cpu_total_time += end_time - start_time;
    // }

    // double cpu_avg_time = cpu_total_time / double(iterations);
    
    // printf("CPU average time: %f miliseconds\n", (cpu_avg_time * TIME_SCALE_MS));

    // // Verify correctness
    // compare_results(h_C, h_C_cpu, M, N);


    //######################### CUBLASTLT FP32 #################################################
    //ALlocate Fp32 matrix
    float *h_C_cublaslt_fp32 = (float*)malloc(size_C);
    float *d_A_fp32, *d_B_fp32, *d_C_fp32;
    CHECK_CUDA(hipMalloc(&d_A_fp32, size_A));
    CHECK_CUDA(hipMalloc(&d_B_fp32, size_B));
    CHECK_CUDA(hipMalloc(&d_C_fp32, size_C));

    //Copy
    CHECK_CUDA(hipMemcpy(d_A_fp32, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_fp32, h_B, size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C_fp32, h_C_cublaslt_fp32, size_C, hipMemcpyHostToDevice));

    //Create cublas handle
    hipblasLtHandle_t handle;
    CHECK_CUBLAS(hipblasLtCreate(&handle));

    //set up matrix descriptors for FP32
    hipblasLtMatrixLayout_t matA_fp32, matB_fp32, matC_fp32;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp32, HIP_R_32F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp32, HIP_R_32F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp32, HIP_R_32F, N, M, N));

    //set up matrix multiplication descriptor for FP32
    hipblasLtMatmulDesc_t matmulDesc_fp32;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp32, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    //set matrix operation for A & B
    hipblasOperation_t transa = HIPBLAS_OP_N; //this is 0 mean no transpose as we already made our matrix has same inner dim
    hipblasOperation_t transb = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(hipblasOperation_t)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(hipblasOperation_t)));

    //setup alpha and beta
    const float alpha = 1.0f;
    const float beta =0.0f;

    //perform multiplication using hipblasLtMatmul Fp32
    CHECK_CUBLAS(hipblasLtMatmul(
        handle,           // (1) Library context handle
        matmulDesc_fp32,  // (2) Operation descriptor
        &alpha,           // (3) Scalar multiplier for operation
        d_B_fp32,         // (4) Input matrix B pointer
        matB_fp32,        // (5) Layout descriptor for matrix B
        d_A_fp32,         // (6) Input matrix A pointer
        matA_fp32,        // (7) Layout descriptor for matrix A
        &beta,            // (8) Scalar multiplier for C
        d_C_fp32,         // (9) Input/output matrix C pointer
        matC_fp32,        // (10) Layout descriptor for C
        d_C_fp32,         // (11) Output matrix D pointer (same as C here)
        matC_fp32,        // (12) Layout descriptor for C
        nullptr,          // (13) Workspace pointer
        nullptr,          // (14) Preferences pointer
        0,                // (15) Workspace size
        0                 // (16) Stream ID
    ));
    
    hipDeviceSynchronize();
    hipMemcpy(h_C_cublaslt_fp32, d_C, size_C, hipMemcpyDeviceToHost);

    bool cublas_fp32_correct = verifyResults(h_C_naive, h_C_cublaslt_fp32, 1e-2, size_C);

    free(h_C_cublaslt_fp32);
    hipFree(d_A_fp32);hipFree(d_B_fp32);hipFree(d_C_fp32);







    // Free memory
    free(h_A); free(h_B); free(h_C_naive); free(h_C_cpu);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;


}