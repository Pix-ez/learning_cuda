#include "hip/hip_runtime.h"
#include "utils.h"
#include "config.h"




// Kernel declaration
__global__ void naiveMatrixMultiply(float *A, float *B, float *C, int M, int N, int K);
__global__ void tiledMatrixMultiply(float *A, float *B, float *C, int M, int N, int K);
__global__ void gememMatrixMultiply(float *A, float *B, float *C, int M, int N, int K);
int main(){

    //Matrix: C = A x B --> [M x K] * [K * N] = [M x N]
    int M = 8192;
    int K = 1024;
    int N = 4096;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
   

    //Allocate host memory 
    float *h_A = (float*)malloc(size_A);
    float *h_B = (float*)malloc(size_B);
    float *h_C_naive = (float*)malloc(size_C);
    float *h_C_cpu = (float*)malloc(size_C);

    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    // Copy inputs to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // int iterations = 20;

    // Warm-up runs
    printf("Performing warm-up runs...\n");

    const int warmup_runs= 3;
    const int benchmark_runs= 20;



//############################################## Naive MM ##############################################

    // std::cout << h_A[1] << "\t"<< h_A[10] <<std::endl;

    dim3 default_blockDim(16, 16);
    dim3 default_gridDim((N + default_blockDim.x - 1) / default_blockDim.x,
                         (M + default_blockDim.y - 1) / default_blockDim.y);

    float naive_kernel = benchmark_kernel([&]() {
        naiveMatrixMultiply<<<default_gridDim, default_blockDim>>>(d_A, d_B, d_C, M, N, K);
    }, warmup_runs, benchmark_runs);
    std::cout << "Naive CUDA kernel average time: " << naive_kernel << " ms" << std::endl;

    hipMemcpy(h_C_naive, d_C, size_C, hipMemcpyDeviceToHost);


//############################################## GEMEM MM ##############################################

    // std::cout << h_A[1] << "\t"<< h_A[10] <<std::endl;

    // dim3 default_blockDim(16, 16);
    // dim3 default_gridDim((N + default_blockDim.x - 1) / default_blockDim.x,
    //                      (M + default_blockDim.y - 1) / default_blockDim.y);

    float gemem_kernel = benchmark_kernel([&]() {
        gememMatrixMultiply<<<default_gridDim, default_blockDim>>>(d_A, d_B, d_C, M, N, K);
    }, warmup_runs, benchmark_runs);
    std::cout << "Gemem coalsed CUDA kernel average time: " << naive_kernel << " ms" << std::endl;

    hipMemcpy(h_C_naive, d_C, size_C, hipMemcpyDeviceToHost);




//############################################## Tiled kernel ##############################################

    // init_matrix(h_A, M, K);
    // init_matrix(h_B, K, N);
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
    
    
    float tiled_cuda_kernel = benchmark_kernel([&]() {
        tiledMatrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    }, warmup_runs, benchmark_runs);
    std::cout << "Tiled CUDA kernel average time: " << tiled_cuda_kernel << " ms" << std::endl;

    // // Copy result back
    // hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);


//###################### CPU reference #####################################################
    // double cpu_total_time = 0.0;
    // for (int i=0; i<iterations; ++i){
    //     double start_time = get_time();
    //     matmul_cpu(h_A, h_B, h_C_cpu, M, N, K);
    //     double end_time = get_time();
    //     cpu_total_time += end_time - start_time;
    // }

    // double cpu_avg_time = cpu_total_time / double(iterations);
    
    // printf("CPU average time: %f miliseconds\n", (cpu_avg_time * TIME_SCALE_MS));

    // // Verify correctness
    // compare_results(h_C, h_C_cpu, M, N);


//######################### CUBLASTLT FP32 #################################################
    //ALlocate Fp32 matrix
    float *h_C_cublaslt_fp32 = (float*)malloc(size_C);
    float *d_A_fp32, *d_B_fp32, *d_C_fp32;
    CHECK_CUDA(hipMalloc(&d_A_fp32, size_A));
    CHECK_CUDA(hipMalloc(&d_B_fp32, size_B));
    CHECK_CUDA(hipMalloc(&d_C_fp32, size_C));

    //Copy
    // init_matrix(h_A, M, K);
    // init_matrix(h_B, K, N);
    CHECK_CUDA(hipMemcpy(d_A_fp32, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_fp32, h_B, size_B, hipMemcpyHostToDevice));
    //CHECK_CUDA(hipMemcpy(d_C_fp32, h_C_cublaslt_fp32, size_C, hipMemcpyHostToDevice));
    // std::cout << h_A[1] << "\t"<< h_A[10] <<std::endl;
    //Create cublas handle
    hipblasLtHandle_t handle;
    CHECK_CUBLAS(hipblasLtCreate(&handle));

    //set up matrix descriptors for FP32
    hipblasLtMatrixLayout_t matA_fp32, matB_fp32, matC_fp32;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp32, HIP_R_32F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp32, HIP_R_32F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp32, HIP_R_32F, N, M, N));

    //set up matrix multiplication descriptor for FP32
    hipblasLtMatmulDesc_t matmulDesc_fp32;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp32, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    //set matrix operation for A & B
    hipblasOperation_t transa = HIPBLAS_OP_N; //this is 0 mean no transpose as we already made our matrix has same inner dim
    hipblasOperation_t transb = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(hipblasOperation_t)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(hipblasOperation_t)));

    //setup alpha and beta
    const float alpha = 1.0f;
    const float beta =0.0f;


    float cublaslt_fp32_time = benchmark_kernel([&]() {
            CHECK_CUBLAS(hipblasLtMatmul(
            handle,           // (1) Library context handle
            matmulDesc_fp32,  // (2) Operation descriptor
            &alpha,           // (3) Scalar multiplier for operation
            d_B_fp32,         // (4) Input matrix B pointer
            matB_fp32,        // (5) Layout descriptor for matrix B
            d_A_fp32,         // (6) Input matrix A pointer
            matA_fp32,        // (7) Layout descriptor for matrix A
            &beta,            // (8) Scalar multiplier for C
            d_C_fp32,         // (9) Input/output matrix C pointer
            matC_fp32,        // (10) Layout descriptor for C
            d_C_fp32,         // (11) Output matrix D pointer (same as C here)
            matC_fp32,        // (12) Layout descriptor for C
            nullptr,          // (13) Workspace pointer
            nullptr,          // (14) Preferences pointer
            0,                // (15) Workspace size
            0                 // (16) Stream ID
        ));

    }, warmup_runs, benchmark_runs);
    std::cout << "CublasLt FP32 kernel average time: " << cublaslt_fp32_time << " ms" << std::endl;

    
  
    hipMemcpy(h_C_cublaslt_fp32, d_C_fp32, size_C, hipMemcpyDeviceToHost);

    bool cublas_fp32_correct = verifyResults(h_C_naive, h_C_cublaslt_fp32, 1e-2, size_C);
    std::cout << "cuBLAS FP32 results " << (cublas_fp32_correct ? "match" : "do not match") << " the naive kernel results within tolerance of 1e-2." << std::endl;

    free(h_C_cublaslt_fp32);
    hipFree(d_A_fp32);hipFree(d_B_fp32);hipFree(d_C_fp32);


//######################### CUBLASTLT FP16 #################################################
    //ALlocate Fp16 matrix

    size_t half_size_A = M * K * sizeof(half);
    size_t half_size_B = K * N * sizeof(half);
    size_t half_size_C = M * N * sizeof(half);
    half *h_C_cublaslt_fp16 = (half*)malloc(half_size_C);
    half *d_A_fp16, *d_B_fp16, *d_C_fp16;
    CHECK_CUDA(hipMalloc(&d_A_fp16, half_size_A));
    CHECK_CUDA(hipMalloc(&d_B_fp16, half_size_B));
    CHECK_CUDA(hipMalloc(&d_C_fp16, half_size_C));

    //first convert to FP16 then copy
    half *h_A_half = (half*)malloc(half_size_A);
    half *h_B_half = (half*)malloc(half_size_B);
    

    for (int i=0; i< M*K; ++i) h_A_half[i] = __float2half(h_A[i]);
    for (int i=0; i< N*K; ++i) h_B_half[i] = __float2half(h_B[i]);

    CHECK_CUDA(hipMemcpy(d_A_fp16, h_A_half, half_size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_fp16, h_B_half, half_size_B, hipMemcpyHostToDevice));
    

    //Create cublas handle
    //hipblasLtHandle_t handle;
    CHECK_CUBLAS(hipblasLtCreate(&handle));

    //set up matrix descriptors for FP16
    hipblasLtMatrixLayout_t matA_fp16, matB_fp16, matC_fp16;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp16, HIP_R_16F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp16, HIP_R_16F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp16, HIP_R_16F, N, M, N));

    //set up matrix multiplication descriptor for FP32
    hipblasLtMatmulDesc_t matmulDesc_fp16;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp16, HIPBLAS_COMPUTE_16F, HIP_R_16F));

    //set matrix operation for A & B
    //hipblasOperation_t transa = HIPBLAS_OP_N; //this is 0 mean no transpose as we already made our matrix has same inner dim
    //hipblasOperation_t transb = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp16, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(hipblasOperation_t)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp16, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(hipblasOperation_t)));

    //setup alpha and beta
    half alpha_half = __float2half(1.0f);
    half beta_half = __float2half(0.0f);

    float cublaslt_fp16_time = benchmark_kernel([&]() {
        CHECK_CUBLAS(hipblasLtMatmul(
            handle,           // (1) Library context handle
            matmulDesc_fp16,  // (2) Operation descriptor
            &alpha_half,           // (3) Scalar multiplier for operation
            d_B_fp16,         // (4) Input matrix B pointer
            matB_fp16,        // (5) Layout descriptor for matrix B
            d_A_fp16,         // (6) Input matrix A pointer
            matA_fp16,        // (7) Layout descriptor for matrix A
            &beta_half,            // (8) Scalar multiplier for C
            d_C_fp16,         // (9) Input/output matrix C pointer
            matC_fp16,        // (10) Layout descriptor for C
            d_C_fp16,         // (11) Output matrix D pointer (same as C here)
            matC_fp16,        // (12) Layout descriptor for C
            nullptr,          // (13) Workspace pointer
            nullptr,          // (14) Preferences pointer
            0,                // (15) Workspace size
            0                 // (16) Stream ID
        ));

    }, warmup_runs, benchmark_runs);
    std::cout << "CublasLt FP16 kernel average time: " << cublaslt_fp16_time << " ms" << std::endl;


    hipMemcpy(h_C_cublaslt_fp16, d_C_fp16, half_size_C, hipMemcpyDeviceToHost);
    for (int i=0; i<M*N; ++i) h_C_cpu[i] = __half2float(h_C_cublaslt_fp16[i]);

    bool cublas_fp16_correct = verifyResults(h_C_naive, h_C_cpu, 1e-2, size_C);
    std::cout << "cuBLAS FP32 results " << (cublas_fp16_correct ? "match" : "do not match") << " the naive kernel results within tolerance of 1e-2." << std::endl;

    free(h_C_cublaslt_fp16);
    hipFree(d_A_fp16);hipFree(d_B_fp16);hipFree(d_C_fp16);

//######################### CUBLASTLT Tensor FP32 #################################################
   
    // Allocate device memory
    float *h_C_cublaslt_tensor_fp32 = (float*)malloc(size_C);
    float *d_A_tensor_fp32, *d_B_tensor_fp32, *d_C_tensor_fp32;
    CHECK_CUDA(hipMalloc(&d_A_tensor_fp32, size_A));
    CHECK_CUDA(hipMalloc(&d_B_tensor_fp32, size_B));
    CHECK_CUDA(hipMalloc(&d_C_tensor_fp32, size_C));

    // Copy inputs
    CHECK_CUDA(hipMemcpy(d_A_tensor_fp32, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_tensor_fp32, h_B, size_B, hipMemcpyHostToDevice));
 

    hipDeviceProp_t deviceProp;
    int deviceId;
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&deviceProp, deviceId);

    // Create cuBLASLt handle
    hipblasLtHandle_t ltHandle;
    CHECK_CUBLAS(hipblasLtCreate(&ltHandle));
    

    

    // Create operation descriptor
    hipblasLtMatmulDesc_t operationDesc;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F_FAST_TF32, HIP_R_32F));

    // Set transposition (no transpose A and B)
    hipblasOperation_t transA = HIPBLAS_OP_T;
    hipblasOperation_t transB = HIPBLAS_OP_T;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB)));

    // Create matrix layouts - CRITICAL: Adjust leading dimensions for transposed matrices
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    // For transposed A (HIPBLAS_OP_T): rows=K, cols=M, leading dimension=K
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, K, M, K));
    // For transposed B (HIPBLAS_OP_T): rows=N, cols=K, leading dimension=N
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, N, K, N));
    // Result C: rows=M, cols=N, leading dimension=M
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, M, N, M));

    // Set alpha and beta
    // float alpha = 1.0f;
    // float beta = 0.0f;

    // Allocate workspace
    void *d_workspace = nullptr;
    size_t workspaceSize = 32 * 1024 * 1024; // 4MB
    CHECK_CUDA(hipMalloc(&d_workspace, workspaceSize));

    // Create preference object
    hipblasLtMatmulPreference_t preference;
    CHECK_CUBLAS(hipblasLtMatmulPreferenceCreate(&preference));
    CHECK_CUBLAS(hipblasLtMatmulPreferenceSetAttribute(preference,
                HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                &workspaceSize, sizeof(workspaceSize)));


    // Force Tensor Core algorithm selection
    int algoMode = 1;  // Tensor Core algorithms only
    CHECK_CUBLAS(hipblasLtMatmulPreferenceSetAttribute(
                preference,
                HIPBLASLT_MATMUL_PREF_SEARCH_MODE,
                &algoMode, sizeof(algoMode)));

    // Create operation descriptor with explicit TF32 computation

    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F_FAST_TF32, HIP_R_32F));
    // Set bias operation to no-op if not needed
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(
                operationDesc,
                HIPBLASLT_MATMUL_DESC_EPILOGUE,
                &epilogue, sizeof(epilogue)));


    // Find the best heuristic algo
    // Request multiple algorithm options and select best performing
    const int REQUEST_ALGO_COUNT = 10;  // Request more algorithm options
    hipblasLtMatmulHeuristicResult_t heuristicResults[REQUEST_ALGO_COUNT];
    int returnedResults = 0;
    CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(
        ltHandle,
        operationDesc,
        Adesc,
        Bdesc,
        Cdesc,
        Cdesc,
        preference,
        REQUEST_ALGO_COUNT,
        heuristicResults,
        &returnedResults
    ));


    // Validate results
    std::cout << "Number of algorithms found: " << returnedResults << std::endl;

    // Select highest performing algorithm (lowest compute time expected)
    int selectedAlgo = 0;
    if (returnedResults > 0) {
        for (int i = 0; i < returnedResults; i++) {
            if (heuristicResults[i].state == HIPBLAS_STATUS_SUCCESS) {
                std::cout << "Algo " << i << " - wavesCount: " << heuristicResults[i].wavesCount;
                std::cout << ", workspaceSize: " << heuristicResults[i].workspaceSize << std::endl;
                selectedAlgo = i;  // Can implement more sophisticated selection criteria
            }
        }
    } else {
        std::cerr << "No valid algorithms found!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Ensure synchronization before timing
    hipDeviceSynchronize();

    // Launch
    float cublaslt_tensor_f32_time = benchmark_kernel([&]() {
        CHECK_CUBLAS(hipblasLtMatmul(ltHandle,
            operationDesc,
            &alpha,
            d_A_tensor_fp32, Adesc,
            d_B_tensor_fp32, Bdesc,
            &beta,
            d_C_tensor_fp32, Cdesc,
            d_C_tensor_fp32, Cdesc,
            &heuristicResults[selectedAlgo].algo,
            d_workspace, workspaceSize,
            0));  // default stream
    // Force synchronization within benchmark loop
    hipDeviceSynchronize();
    }, warmup_runs, benchmark_runs);

    

    std::cout << "cuBLASLt Tensor FP32 (Tensor Core) avg time: " << cublaslt_tensor_f32_time << " ms" << std::endl;

    // Cleanup
    CHECK_CUBLAS(hipblasLtMatmulPreferenceDestroy(preference));
    CHECK_CUDA(hipFree(d_workspace));
    CHECK_CUBLAS(hipblasLtDestroy(ltHandle));

    // float cublaslt_tensor_f32_time = benchmark_kernel([&]() {
    //     CHECK_CUBLAS(hipblasLtMatmul(ltHandle,
    //         operationDesc,
    //         &alpha,
    //         d_A_tensor_fp32, Adesc,
    //         d_B_tensor_fp32, Bdesc,
    //         &beta,
    //         d_C_tensor_fp32, Cdesc,
    //         d_C_tensor_fp32, Cdesc,
    //         nullptr, nullptr, 0, 0););

    // }, warmup_runs, benchmark_runs);
    // std::cout << "CublasLt Tensor FP32 kernel average time: " << cublaslt_tensor_f32_time << " ms" << std::endl;


    hipMemcpy(h_C_cublaslt_tensor_fp32, d_C_tensor_fp32, size_C, hipMemcpyDeviceToHost);

    bool cublas_tensor_fp32_correct = verifyResults(h_C_naive, h_C_cublaslt_tensor_fp32, 1e-2, size_C);
    std::cout << "cuBLAS Tensor FP32 results " << (cublas_tensor_fp32_correct ? "match" : "do not match") << " the naive kernel results within tolerance of 1e-2." << std::endl;

    free(h_C_cublaslt_tensor_fp32);
    hipFree(d_A_tensor_fp32);hipFree(d_B_tensor_fp32);hipFree(d_C_tensor_fp32);



    // Free memory
    free(h_A); free(h_B); free(h_C_naive); free(h_C_cpu);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;


}